#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <fstream>
#include <numeric>

#include "../include/literal.cu"
#include "../include/cuda_error.cu"
#include "../include/cuda_utilities.cu"

__device__ __host__ void print_array(int* array, int length, int values_per_row, const char* row_prefix) {
    for (int i = 0; i < length; ++i) {
        if (i % values_per_row == 0) {
            printf("%s", row_prefix);
        }
        printf("%d ", array[i]);
        if ((i + 1) % values_per_row == 0) {
            printf("\n");
        }
    }
    if (length % values_per_row != 0) {
        printf("\n");
    }
}
__device__ __host__ void print_array(bool* array, int length, int values_per_row, const char* row_prefix = "") {
    for (int i = 0; i < length; ++i) {
        if (i % values_per_row == 0) {
            printf("%s", row_prefix);
        }
        printf("%d ", array[i]);
        if ((i + 1) % values_per_row == 0) {
            printf("\n");
        }
    }
    if (length % values_per_row != 0) {
        printf("\n");
    }
}

__device__ __host__ void print_array(int* array, int num_rows, int* row_end_indices, const char* row_prefix = "") {
    int start_index = 0;
    for (int row = 0; row < num_rows; ++row) {
        printf("%s", row_prefix);
        for (int index = start_index; index < row_end_indices[row]; ++index) {
            printf("%d ", array[index]);
        }
        printf("\n");
        start_index = row_end_indices[row];
    }
}

/*
Runs 1 block per candidate solution.
Each block loads the value of the j-th component of the candidate solution and its influence on the other components.
Then, it propagates the influence of the j-th component to all other components.
Repeats for all components of the candidate solution, starting from the last one.

Args:
    n_comp: number of components.
    n_sol: number of candidate solutions.
    n_vars: number of variables.
    candidates: matrix of candidate solutions.
        each row contains a valid permutation of the sccs.
    infl_comp: matrix of influence between components.
        each row contains the influence of the i-th component due to negated variables.
    comp: vertex component mapping.
    sol_comp: matrix of solutions on components, must be initialized with all values = true,
        it will contain the results.
*/
__global__ void kernel_solve_2SAT(int n_comp, int n_sol, int n_vars, int* candidates, int* comp, int* infl_comp, int* infl_comp_end_idx, bool* sol_comp) {
    __shared__ int curr_comp;
    __shared__ bool val_i;
    __shared__ int* infl_i;
    __shared__ int infl_i_size;
    __shared__ bool* sol_i;
    for (int i = 0; i < n_sol; i += gridDim.x) {
        int curr_sol = blockIdx.x + i;
        if (curr_sol >= n_sol) return;
        for (int j = n_comp-1; j >= 0; --j) {
            // load in shared mem the j-th component of the current candidate solution
            if (threadIdx.x == 0) {
                curr_comp = candidates[curr_sol * n_comp + j];
                sol_i = sol_comp + (curr_sol * n_comp);
                int offset = curr_comp == 0 ? 0 : infl_comp_end_idx[curr_comp - 1];
                infl_i_size = infl_comp_end_idx[curr_comp] - offset;
                infl_i = infl_comp + offset;
                val_i = sol_i[curr_comp];
            }
            __syncthreads();
            // propagate the effect of the j-th component to all other components
            for (int k = 0; k < infl_i_size; k += blockDim.x) {
                int infl_idx = threadIdx.x + k;
                if (infl_idx >= infl_i_size) return;
                int target_comp = infl_i[infl_idx];
                sol_i[target_comp] = !val_i;
            }
            __syncthreads();
        }
    }
}

void compute_sccs_solutions(int max_threads, int max_blocks, int n_comp, int n_sol, int n_vars, int n_vertices,
                            int* h_candidates, int* h_infl_comp, int* h_infl_comp_end_idx, size_t infl_comp_bytes, int* h_comp,
                            int** d_comp, bool** d_sol_comp) {
    int threads_per_block = std::min(max_threads, n_comp);
    int n_blocks = std::min(max_blocks, n_sol);

    bool* h_sol_comp = (bool*)malloc(n_sol * n_comp * sizeof(bool));
    std::fill(h_sol_comp, h_sol_comp + n_sol * n_comp, true);
    
    size_t candidates_size = n_sol * n_comp * sizeof(int);
    size_t comp_size = n_vertices * sizeof(int);
    size_t sol_comp_size = n_sol * n_comp * sizeof(bool);
    size_t infl_comp_sizes_size = n_comp * sizeof(int);
    size_t max_heap_size = candidates_size + infl_comp_bytes + infl_comp_sizes_size + comp_size + sol_comp_size;
    set_heap_size(max_heap_size);

    int* d_candidates;
    int* d_infl_comp;
    int* d_infl_comp_sizes;
    HANDLE_ERROR(hipMalloc((void**)&d_candidates, candidates_size));
    HANDLE_ERROR(hipMemcpy(d_candidates, h_candidates, candidates_size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMalloc((void**)&d_infl_comp, infl_comp_bytes));
    HANDLE_ERROR(hipMemcpy(d_infl_comp, h_infl_comp, infl_comp_bytes, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMalloc((void**)&d_infl_comp_sizes, infl_comp_sizes_size));
    HANDLE_ERROR(hipMemcpy(d_infl_comp_sizes, h_infl_comp_end_idx, infl_comp_sizes_size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMalloc((void**)d_comp, comp_size));
    HANDLE_ERROR(hipMemcpy(*d_comp, h_comp, comp_size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMalloc((void**)d_sol_comp, sol_comp_size));
    HANDLE_ERROR(hipMemcpy(*d_sol_comp, h_sol_comp, sol_comp_size, hipMemcpyHostToDevice));

    // printf("Computing sccs solutions...\n");
    // printf("n_blocks: %d, threads_per_block: %d\n", n_blocks, threads_per_block);
    kernel_solve_2SAT<<<n_blocks, threads_per_block>>>(n_comp, n_sol, n_vars, d_candidates, *d_comp, d_infl_comp, d_infl_comp_sizes, *d_sol_comp);
    hipDeviceSynchronize();
    checkCUDAError("computed sccs solutions");
    // printf("Computing sccs solutions done.\n");

    // Copy results back to host
    HANDLE_ERROR(hipMemcpy(h_sol_comp, *d_sol_comp, n_sol * n_comp * sizeof(bool), hipMemcpyDeviceToHost));
    // print_array(h_sol_comp, n_sol * n_comp, n_comp);

    // Free device memory
    HANDLE_ERROR(hipFree(d_candidates));
    HANDLE_ERROR(hipFree(d_infl_comp));

    // Free host memory
    free(h_candidates);
    free(h_infl_comp);
    free(h_comp);
}


/*
Runs 1 block per variable.
Each block load the component that contains its variable.
Then, for each candidate solution, it copies the value of the component to the variable.

Args:
    n_comp: number of components.
    n_sol: number of candidate solutions.
    n_vars: number of variables.
    comp: vertex component mapping.
    sol_comp: matrix of solutions on components, it should contain the results already computed.
    sol_var: matrix of solutions on variables, must be initialized with all values = false,
        it will contain the results.
*/
__global__ void kernel_comp_to_var(int n_comp, int n_vars, int n_sol, int* comp, bool* sol_comp, bool* sol_var) {
    __shared__ int var_comp;

    for (int i = 0; i < n_vars; i += gridDim.x) {
        int curr_var = blockIdx.x + i;
        if (curr_var >= n_vars) return;
        // load in shared mem the component of the current variable
        if (threadIdx.x == 0) {
            var_comp = comp[curr_var];
        }
        __syncthreads();
        for (int j = 0; j < n_sol; j += blockDim.x) {
            int curr_sol = threadIdx.x + j;
            if (curr_sol >= n_sol) return;
            sol_var[curr_sol * n_vars + curr_var] = sol_comp[curr_sol * n_comp + var_comp];
        }
        __syncthreads();
    }
}

void solutions_sccs_to_vars(int max_threads, int max_blocks, int n_comp, int n_sol, int n_vars, int n_vertices,
                            int* d_comp, bool* d_sol_comp, bool** h_sol_var) {
    int threads_per_block = std::min(max_threads, n_sol);
    int n_blocks = std::min(max_blocks, n_vars);

    *h_sol_var = (bool*)malloc(n_sol * n_vars * sizeof(bool));
    std::fill(*h_sol_var, *h_sol_var + n_sol * n_vars, false);
    
    size_t sol_var_size = n_sol * n_vars * sizeof(bool);
    size_t comp_size = n_vertices * sizeof(int);
    size_t sol_comp_size = n_sol * n_comp * sizeof(bool);
    size_t max_heap_size = comp_size + sol_comp_size + sol_var_size;
    set_heap_size(max_heap_size);

    bool* d_sol_var;
    HANDLE_ERROR(hipMalloc((void**)&d_sol_var, sol_var_size));

    // printf("Converting sccs solutions to variable solutions...\n");
    kernel_comp_to_var<<<n_blocks, threads_per_block>>>(n_comp, n_vars, n_sol, d_comp, d_sol_comp, d_sol_var);
    hipDeviceSynchronize();
    checkCUDAError("converted sccs solutions to variable solutions");
    // printf("Converting sccs solutions to variable solutions done.\n");

    // Copy results back to host
    HANDLE_ERROR(hipMemcpy(*h_sol_var, d_sol_var, n_sol * n_vars * sizeof(bool), hipMemcpyDeviceToHost));
    // print_array(*h_sol_var, n_sol * n_vars, n_vars);

    // Free device memory
    HANDLE_ERROR(hipFree(d_comp));
    HANDLE_ERROR(hipFree(d_sol_comp));
    HANDLE_ERROR(hipFree(d_sol_var));
}


/*
Runs 1 block per candidate solution pair.
Each block identifies the i-th and j-th candidate solutions, then initializes a counter to 0.
Each thread compares the values of 1 variable of the i-th and j-th candidate solutions.
If the values are different, it increments the counter.
When all threads finish, the block checks if the counter is greater than or equal to min_dist.

Args:
    n_sol: number of candidate solutions.
    n_vars: number of variables.
    sol_var: matrix of solutions on variables, it should contain the results already computed.
    sol_var_min_dist: matrix of compatibility between solutions, it should be initialized with all values = false,
        it will contain the results.
    min_dist: minimum hamming distance between two solutions to be considered compatible.
*/
__global__ void kernel_filter_min_dist(int n_sol, int n_vars, bool* sol_var, bool* sol_var_min_dist, int min_dist) {
    __shared__ int counter;
    
    for (int i = 0; i < n_sol * n_sol; i += gridDim.x) {
        int curr_sol_row = (blockIdx.x + i) / n_sol;
        if (curr_sol_row >= n_sol) return;
        int curr_sol_col = (blockIdx.x + i) % n_sol;
        if (curr_sol_col >= n_sol) return; // cannot happen

        if (curr_sol_row < curr_sol_col) return; // skip symmetric pairs
        if (curr_sol_row == curr_sol_col) return; // skip diagonal

        if (threadIdx.x == 0) {
            counter = 0;
        }
        __syncthreads();

        for (int j = 0; j < n_vars; j += blockDim.x) {
            int curr_var = threadIdx.x + j;
            if (curr_var >= n_vars) return;
            
            if (sol_var[curr_sol_row * n_vars + curr_var] != sol_var[curr_sol_col * n_vars + curr_var]) {
                atomicAdd(&counter, 1); // should be at block level, but this version is compatible with all architectures
            }
        }
        __syncthreads();

        if (threadIdx.x == 0) {
            sol_var_min_dist[curr_sol_row * n_sol + curr_sol_col] = counter >= min_dist;
            sol_var_min_dist[curr_sol_col * n_sol + curr_sol_row] = counter >= min_dist;
        }
        __syncthreads();
    }
}

void solutions_hamming_dist(int max_threads, int max_blocks, int n_sol, int n_vars, int min_dist, int ins_sol_size,
                            bool* out_results, bool* h_sol_var, bool** h_sol_var_min_dist) {
    // Append new solutions to the results of last iteration
    int n_sol_var_to_filter = n_sol + ins_sol_size;

    size_t sol_var_to_filter_size = n_sol_var_to_filter * n_vars * sizeof(bool);
    size_t sol_var_min_dist_size = n_sol_var_to_filter * n_sol_var_to_filter * sizeof(bool);
    size_t max_heap_size = sol_var_to_filter_size + sol_var_min_dist_size;
    set_heap_size(max_heap_size);

    bool* h_sol_var_to_filter = (bool*)malloc(n_sol_var_to_filter * n_vars * sizeof(bool));
    memcpy(h_sol_var_to_filter, out_results, ins_sol_size * n_vars * sizeof(bool));
    memcpy(h_sol_var_to_filter + ins_sol_size * n_vars, h_sol_var, n_sol * n_vars * sizeof(bool));

    bool* d_sol_var_to_filter;
    HANDLE_ERROR(hipMalloc((void**)&d_sol_var_to_filter, sol_var_to_filter_size));
    HANDLE_ERROR(hipMemcpy(d_sol_var_to_filter, h_sol_var_to_filter, sol_var_to_filter_size, hipMemcpyHostToDevice));

    int threads_per_block = std::min(max_threads, n_vars);
    int n_blocks = std::min(max_blocks, n_sol_var_to_filter * n_sol_var_to_filter);

    bool* d_sol_var_min_dist;
    HANDLE_ERROR(hipMalloc((void**)&d_sol_var_min_dist, sol_var_min_dist_size));

    // printf("Computing compatibility between solutions...\n");
    kernel_filter_min_dist<<<n_blocks, threads_per_block>>>(n_sol_var_to_filter, n_vars, d_sol_var_to_filter, d_sol_var_min_dist, min_dist);
    hipDeviceSynchronize();
    checkCUDAError("filtered sccs solutions by min dist");
    // printf("Computing compatibility between solutions done.\n");

    // Copy results back to host
    *h_sol_var_min_dist = (bool*)malloc(n_sol_var_to_filter * n_sol_var_to_filter * sizeof(bool));
    HANDLE_ERROR(hipMemcpy(*h_sol_var_min_dist, d_sol_var_min_dist, n_sol_var_to_filter * n_sol_var_to_filter * sizeof(bool), hipMemcpyDeviceToHost));
    // print_array(*h_sol_var_min_dist, n_sol_var_to_filter * n_sol_var_to_filter, n_sol_var_to_filter);

    // Free device memory
    HANDLE_ERROR(hipFree(d_sol_var_to_filter));
    HANDLE_ERROR(hipFree(d_sol_var_min_dist));

    // Free host memory
    free(h_sol_var_to_filter);
}


/*
Serial solution.
Builds a vector of indices of the candidate solutions already inserted in the output pool. The indices refer to the h_sol_var_min_dist matrix.
Then, for each candidate solution in the new pool, it checks if it is compatible with all the solutions already inserted in the output pool.
If it is compatible, it is inserted in the output pool.

Args:
    init: true if this is the first iteration, false otherwise.
    n_sol: number of candidate solutions.
    n_vars: number of variables.
    n: maximum number of solutions to output.
    h_sol_var: matrix of new solutions on variables, it should contain the results already computed.
    h_sol_var_min_dist: matrix of compatibility between solutions, contains the compatibility between all candidates in the output pool and in the new pool.
    out_results: matrix of final results.
    n_out_results: number of candidate solutions already in out_results.
*/
int insert_new_solution(bool init, int n_sol, int n_vars, int n, bool* h_sol_var, bool* h_sol_var_min_dist,
    bool* out_results, int n_out_results) {
    // printf("Building final results...\n");
    int n_sol_var_to_filter = n_sol + n_out_results;
    std::vector<int> inserted_solutions(n_out_results);
    std::iota(inserted_solutions.begin(), inserted_solutions.end(), 0);

    // insert the first result
    if (init) {
    memcpy(out_results, h_sol_var, n_vars * sizeof(bool));
    inserted_solutions.push_back(0);
    }

    // insert the rest of the results
    for (int i = inserted_solutions.size(); i < n_sol_var_to_filter && inserted_solutions.size() < n; ++i) {
        bool valid = true;
        for (int val : inserted_solutions) {
            if (!h_sol_var_min_dist[i * n_sol_var_to_filter + val]) {
                valid = false;
                break;
            }
        }
        if (valid) {
            memcpy(out_results + inserted_solutions.size() * n_vars,
            h_sol_var + (i-n_out_results) * n_vars, n_vars * sizeof(bool));
            inserted_solutions.push_back(i);
        }
    }

    // Free host memory
    free(h_sol_var);
    free(h_sol_var_min_dist);

    // printf("Building final results done.\n");
    return inserted_solutions.size();
}

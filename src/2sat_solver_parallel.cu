#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cassert>
#include <algorithm>
#include <fstream>

#include "../include/literal.cu"
#include "../include/cuda_error.cu"

void add_disjunction(Literal var1, Literal var2, bool** adj, bool** adj_t) {
    unsigned int a = var1.value;
    unsigned int b = var2.value;
    bool na = !var1.isPositive;
    bool nb = !var2.isPositive;
    // Note: remember, the k-th variable is represented by 2 * k and 2 * k + 1 (its negation) 
    // an even number has always the LSB set to 0, so the XOR with 1 is equivalent to adding 1
    a = 2 * a ^ na;
    b = 2 * b ^ nb;
    // Note2: if a variable is negated, then its LSB is set to 1, so the XOR with 1 is equivalent to subtracting 1
    // otherwise, the XOR with 1 is equivalent to adding 1
    int neg_a = a ^ 1;
    int neg_b = b ^ 1;
    adj[neg_a][b] = true;
    adj[neg_b][a] = true;
    adj_t[b][neg_a] = true;
    adj_t[a][neg_b] = true;
}

int fill_adjacency_matrix(std::string filepath, bool*** adj, bool*** adj_t) {
    std::ifstream file(filepath);
    std::vector<Literal> vars;
    std::string var1, var2;
    while (file >> var1 >> var2) {
        vars.push_back(Literal(var1));
        vars.push_back(Literal(var2));
    }

    Literal max_var = *max_element(vars.begin(), vars.end());
    size_t n_vars = max_var.value + 1;
    size_t n_vertices = 2 * n_vars;
    *adj = (bool**)malloc(n_vertices * sizeof(bool*));
    *adj_t = (bool**)malloc(n_vertices * sizeof(bool*));
    for (size_t i = 0; i < n_vertices; ++i) {
        (*adj)[i] = (bool*)malloc(n_vertices * sizeof(bool));
        (*adj_t)[i] = (bool*)malloc(n_vertices * sizeof(bool));
        for (size_t j = 0; j < n_vertices; ++j) {
            (*adj)[i][j] = false;
            (*adj_t)[i][j] = false;
        }
    }
    for (size_t i = 0; i < vars.size(); i += 2) {
        add_disjunction(vars[i], vars[i + 1], *adj, *adj_t);
    }
    return n_vertices;
}

__device__ void dfs1(int v, int n_vertices, bool* used, int* order, int order_count, bool** adj) {
    used[v] = true;
    for (int u = 0; u < n_vertices; ++u) {
        if (adj[v][u] && !used[u])
            dfs1(u, n_vertices, used, order, order_count, adj);
    }
    order[order_count++] = v;
}

__device__ void dfs2(int v, int cl, int n_vertices, int* comp, bool** adj_t) {
    comp[v] = cl;
    for (int u = 0; u < n_vertices; ++u) {
        if (adj_t[v][u] && comp[u] == -1)
            dfs2(u, cl, n_vertices, comp, adj_t);
    }
}

__device__ bool solve_2SAT(int n_vars, int n_vertices, bool* used, int* order, int* comp, bool** adj, bool** adj_t, bool* assignment, int start_node = 0) {
    for (int i = 0; i < n_vertices; ++i) {
        order[i] = -1;
        used[i] = false;
        comp[i] = -1;
    }
    // prepare the dfs order starting from the specified node
    dfs1(start_node, n_vertices, used, order, 0, adj);
    for (int i = 0; i < n_vertices; ++i) {
        if (!used[i]) // handle the case where the graph is not connected
            dfs1(i, n_vertices, used, order, 0, adj);
    }

    // identify the strongly connected components and create a topological order
    for (int i = 0, j = 0; i < n_vertices; ++i) {
        int v = order[n_vertices - i - 1];
        if (comp[v] == -1)
            dfs2(v, j++, n_vertices, comp, adj_t);
    }

    for (int i = 0; i < n_vars; ++i) {
        assignment[i] = false;
    }
    // check if the 2-SAT problem is satisfiable
    for (int i = 0; i < n_vertices; i += 2) {
        if (comp[i] == comp[i + 1]) // if a variable and its negation are in the same strongly connected component
            return false;
        assignment[i / 2] = comp[i] > comp[i + 1];
    }
    return true;
}


__global__ void kernel_solve_2SAT(bool** results, bool* solvable, int start_node, int n_vars, bool** adj, bool** adj_t) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int current_vertex = tid + start_node;
    int n_vertices = 2 * n_vars;
    if (current_vertex < n_vertices) {
        bool* assignment = (bool*)malloc(n_vars * sizeof(bool));
        int* order = (int*)malloc(n_vertices * sizeof(int));
        int* comp = (int*)malloc(n_vertices * sizeof(int));
        bool* used = (bool*)malloc(n_vertices * sizeof(bool));
        if (solve_2SAT(n_vars, n_vertices, used, order, comp, adj, adj_t, assignment, current_vertex)) {
            solvable[current_vertex] = true;
            for (int i = 0; i < n_vars; ++i) {
                results[current_vertex][i] = assignment[i];
            }
        } else {
            solvable[current_vertex] = false;
        }
        free(assignment);
        free(order);
        free(comp);
        free(used);
    }
}
#include "2sat_solver_linear.cu"
#include "2sat_solver_parallel.cu"

#include "../include/cuda_error.cu"
#include "../include/cuda_utilities.cu"

void linear_usage(std::string filename) {
    TwoSatSolverLinear solver_lin = TwoSatSolverLinear(filename);
    if (!solver_lin.solve_2SAT()) {
        std::cout << "No solution" << std::endl;
        return;
    }
    solver_lin.solve_from_all_nodes(1000, 10);
    for (const auto& sol : solver_lin.solutions) {
        std::cout << "solution: ";
        for (bool val : sol) {
            std::cout << val << " ";
        }
        std::cout << std::endl;
    }
}

void parallel_usage(std::string filename) {
    // initialize adjacency matrix
    bool **h_adj, **h_adj_t;
    bool **d_adj, **d_adj_t;
    int n_vertices = fill_adjacency_matrix(filename, &h_adj, &h_adj_t);
    int n_vars = n_vertices / 2;


    std::cout << "Number of vertices: " << n_vertices << std::endl;
    // allocate in global memory
    HANDLE_ERROR(hipMalloc(&d_adj, n_vertices * sizeof(bool*)));
    HANDLE_ERROR(hipMalloc(&d_adj_t, n_vertices * sizeof(bool*)));
    for (int i = 0; i < n_vertices; ++i) {
        std::cout << "Allocating " << i << std::endl;
        HANDLE_ERROR(hipMalloc((void**)&d_adj[i], n_vertices * sizeof(bool)));
        std::cout << "Allocating " << i << std::endl;
        HANDLE_ERROR(hipMemcpy(d_adj[i], h_adj[i], n_vertices * sizeof(bool), hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMalloc(&d_adj_t[i], n_vertices * sizeof(bool)));
        HANDLE_ERROR(hipMemcpy(d_adj_t[i], h_adj_t[i], n_vertices * sizeof(bool), hipMemcpyHostToDevice));
    }
    
    int threads_per_block = get_device_prop(0).maxThreadsPerBlock;
    int n = 10;

    std::cout << "Solving 2SAT in linear" << std::endl;

    // initialize results matrix
    bool **d_results, *d_solvable;
    HANDLE_ERROR(hipMalloc(&d_solvable, n * sizeof(bool)));
    HANDLE_ERROR(hipMalloc(&d_results, n * sizeof(bool*)));
    for (int i = 0; i < n; ++i) {
        HANDLE_ERROR(hipMalloc(&d_results[i], n_vars * sizeof(bool)));
    }

    printf("Solving 2SAT in parallel\n");
    kernel_solve_2SAT<<<(n + threads_per_block - 1) / threads_per_block, threads_per_block>>>(d_results, d_solvable, 0, n_vars, d_adj, d_adj_t);
    hipDeviceSynchronize();
    checkCUDAError("parallel 2SAT solver");
}

int main(int argc, char** argv) {
    if (argc < 2) {
        std::cout << "Usage: " << argv[0] << " <filename>" << std::endl;
        return 1;
    }
    const char* filename = argv[1];
    
    linear_usage(filename);
    parallel_usage(filename);

    return 0;
}

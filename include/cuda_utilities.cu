#ifndef __CUDA_UTILITIES_CU__
#define __CUDA_UTILITIES_CU__

#include <stdio.h>
#include "./cuda_error.cu"    

hipDeviceProp_t get_device_prop(int i) {
    hipDeviceProp_t prop;
    HANDLE_ERROR( hipGetDeviceProperties( &prop, i ) );
    return prop;
}

void print_prop_summary(int i) {
    hipDeviceProp_t prop = get_device_prop(i);
    printf( " --- General Information for device %d ---\n", i );
    printf( "Name: %s\n", prop.name );
    printf( "Compute capability: %d.%d\n", prop.major, prop.minor );
    printf( "Clock rate: %d\n", prop.clockRate );
    printf( "Device copy overlap: " );
    if (prop.deviceOverlap) printf( "Enabled\n" );
    else printf( "Disabled\n");
    printf( "Kernel execution timeout : " ); 
    if (prop.kernelExecTimeoutEnabled) printf( "Enabled\n" );
    else printf( "Disabled\n" );
    printf( " --- Memory Information for device %d ---\n", i );
    printf( "Total global mem: %ld\n", prop.totalGlobalMem );
    printf( "Total constant Mem: %ld\n", prop.totalConstMem );
    printf( "Max mem pitch: %ld\n", prop.memPitch );
    printf( "Texture Alignment: %ld\n", prop.textureAlignment );
    printf( " --- MP Information for device %d ---\n", i );
    printf( "Multiprocessor count: %d\n", prop.multiProcessorCount );
    printf( "Shared mem per mp: %ld\n", prop.sharedMemPerBlock );
    printf( "Registers per mp: %d\n", prop.regsPerBlock );
    printf( "Threads in warp: %d\n", prop.warpSize );
    printf( "Max threads per block: %d\n", prop.maxThreadsPerBlock );
    printf( "Max thread dimensions: (%d, %d, %d)\n",
    prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2] );
    printf( "Max grid dimensions: (%d, %d, %d)\n",
    prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2] );
    printf( "\n" );
}

void set_heap_size(size_t max_heap_size) {
    size_t free_mem, total_mem;
    HANDLE_ERROR(hipMemGetInfo(&free_mem, &total_mem));
    if (max_heap_size > free_mem) {
        std::cerr << "Not enough memory on device. Required: " << max_heap_size / (1024 * 1024) << " MB, Available: " << free_mem / (1024 * 1024) << " MB" << std::endl;
        return;
    }
    hipDeviceSetLimit(hipLimitMallocHeapSize, max_heap_size);
    checkCUDAError("Heap size set");
}

#endif // __CUDA_UTILITIES_CU__

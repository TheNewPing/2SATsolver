#ifndef __CUDA_ERROR_CU__
#define __CUDA_ERROR_CU__

#include <stdio.h>

static void HandleError(hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line ); 
        exit( EXIT_FAILURE );
    }
}
       
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError(); 
    if( hipSuccess != err) {
fprintf(stderr, "CUDA ERROR: >%s<: >%s<. Executing: EXIT\n", msg, hipGetErrorString(err) ); exit(-1);
    }
}

#endif // __CUDA_ERROR_CU__